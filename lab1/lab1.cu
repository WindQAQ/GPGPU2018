#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdint>
#include <algorithm>
#include "lab1.h"

#define dt 0.5f
#define RGB2Y(r, g, b) ( +0.299 * (r) + 0.587 * (g) + 0.114 * (b) )
#define RGB2U(r, g, b) ( -0.169 * (r) - 0.331 * (g) + 0.500 * (b) + 128.0 )
#define RGB2V(r, g, b) ( +0.500 * (r) - 0.419 * (g) - 0.081 * (b) + 128.0 )

static const unsigned W = 512;
static const unsigned H = 512;
static const unsigned NFRAME = 240;
static const int SIZE = W *H;
static const int THREADS_PER_BLOCK = 256;
static const int NUMBER_OF_BLOCKS = SIZE / THREADS_PER_BLOCK;
static const int NUMBER_OF_ITERATIONS = 50;

enum class Boundary {
	U, 
	V, 
	D
};

__device__ inline bool not_boundary(int x, int y)
{
	// the boundary value is the `wall`
	return (x > 0) && (y > 0) && (x < W - 1) && (y < H - 1);
}

__device__ inline float clamp(float x, float _min, float _max)
{
	return fminf(fmaxf(x, _min), _max);
}

__device__ inline float bilinear_interpolate(float x, float y, const float *s)
{
	int px0, px1, py0, py1;
	float dx0, dx1, dy0, dy1;

	px0 = __float2int_rd(x), py0 = __float2int_rd(y);
	px1 = px0 + 1, py1 = py0 + 1;
	dx1 = x - px0, dy1 = y - py0;
	dx0 = 1.0 - dx1; dy0 = 1.0 - dy1;

	return dx0 * (dy1 * s[px0 + W * py0] + dy1 * s[px0 + W * py1]) +
			dx1 * (dy0 * s[px1 + W * py0] + dy1 * s[px1 + W * py1]);
}

__global__ void density_to_yuv(const float *d, uint8_t *yuv)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	const float r = 0.0f,
				g = 0.0f,
				b = clamp(d[idx], 0.0f, 255.0f);

	yuv[idx] = static_cast<uint8_t>(RGB2Y(r, g, b));

	if (x % 2 == 1 && y % 2 == 1) {
		const int sx = x / 2, sy = y / 2;
		const int sidx = sx + sy * W / 2;
		yuv[SIZE + sidx] = static_cast<uint8_t>(RGB2U(r, g, b));
		yuv[SIZE + SIZE / 4 + sidx] = static_cast<uint8_t>(RGB2V(r, g, b));
	}
}

__global__ void add_force(float *d, const float *s)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		d[idx] += dt * s[idx];
	}
}

__global__ void transport(float *d, const float *d0, const float *u, const float *v, const float dissipation)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	float nx, ny;

	if (not_boundary(x, y)) {
		// trace practicle
		nx = x - dt * u[idx];
		ny = y - dt * v[idx];

		// clip value out of boundary
		nx = clamp(nx, 0.5, W - 1.5);
		ny = clamp(ny, 0.5, H - 1.5);

		// bi-linear interpolate
		d[idx] = bilinear_interpolate(nx, ny, d0);
		d[idx] *= dissipation;
	}
}

__global__ void get_divergence(const float *u, const float *v, float *div)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		// calculate gradient from neighbors (difference)
		div[idx] = -0.5 * ( (u[idx + 1] - u[idx - 1]) +
					(v[x + (y + 1) * W] - v[x + (y - 1) * W]) );
	}
}

__global__ void linear_solve(const float *u, const float *v, const float *div, const float *p0, float *p)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		p[idx] = (p0[idx - 1] + p0[idx + 1] + p0[x + (y - 1) * W] + 
					p0[x + (y + 1) * W] + div[idx]) * 0.25;
	}
}

__global__ void update_velocity(float *u, float *v, const float *p)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		u[idx] -= 0.5 * (p[idx + 1] - p[idx - 1]);
		v[idx] -= 0.5 * (p[x + (y + 1) * W] - p[x + (y - 1) * W]);
	}
}

__global__ void set_boundary(float *p, Boundary mode)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (x == 0) {
		switch (mode) {
			case Boundary::U: p[idx] = -p[idx + 1]; break;
			case Boundary::V: case Boundary::D: p[idx] = p[idx + 1]; break;
		}
	}
	else if (y == 0) {
		switch (mode) {
			case Boundary::U: case Boundary::D: p[idx] = p[x + (y + 1) * W]; break;
			case Boundary::V: p[idx] = -p[x + (y + 1) * W]; break;
		}
	}
	else if (x == W - 1) {
		switch (mode) {
			case Boundary::U: p[idx] = -p[idx - 1]; break;
			case Boundary::V: case Boundary::D: p[idx] = p[idx - 1]; break;
		}		
	}
	else if (y == H - 1) {
		switch (mode) {
			case Boundary::U: case Boundary::D: p[idx] = p[x + (y - 1) * W]; break;
			case Boundary::V: p[idx] = -p[x + (y - 1) * W]; break;
		}
	}
}

__global__ void init_source(float *u, float *v, float *d)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	int dx = x - W / 2, dy = -(y - H / 2);
	int distance = dx * dx + dy * dy;

	// if (distance <= 2500) {
	// 	u[idx] = 10.0 * (dy - dx);
	// 	v[idx] = 10.0 * (dx + dy);
	// }

	u[idx] = 50.0;

	dx = x - W / 2 + 100, dy = -(y - H / 2);
	distance = dx * dx + dy * dy;

	if (distance <= 900)  d[idx] = 100.0;

	dx = x - W / 2 - 100, dy = -(y - H / 2);
	distance = dx * dx + dy * dy;

	if (distance <= 900)  d[idx] = 100.0;
}

__global__ void compute_curl(const float *u, const float *v, float *curl)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		curl[idx] = fabsf(0.5 * (v[idx + 1] - v[idx - 1] - u[x + (y + 1) * W] + u[x + (y - 1) * W]));
	}
}

__global__ void vc_update(float *u0, float *v0, const float *curl, const float *u, const float *v)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int x = idx % W, y = idx / W;

	if (not_boundary(x, y)) {
		float dw_dx = (curl[idx + 1] - curl[idx - 1]) * 0.5;
		float dw_dy = (curl[x + (y + 1) * W] - curl[x + (y - 1) * W]) * 0.5;

		float len = sqrtf(dw_dx * dw_dx + dw_dy * dw_dy) + 1e-6;

		dw_dx /= len, dw_dy /= len;

		float c = (u[x + (y + 1) * W] - u[x + (y - 1) * W]) * 0.5 -
					(v[idx + 1] - v[idx - 1]) * 0.5;

		u0[idx] = dw_dy * -c;
		v0[idx] = dw_dx * c;
	}
}

__host__ void project(float *u, float *v, float *p, float *p0, float *div)
{
	get_divergence <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, v, div);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (div, Boundary::D);

	hipMemset(p, 0, SIZE * sizeof(float));
	hipMemset(p0, 0, SIZE * sizeof(float));

	for (int i = 0; i < NUMBER_OF_ITERATIONS; i++) {
		linear_solve <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, v, div, p0, p);
		
		// set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (p, Boundary::D);

		hipMemcpy(p0, p, SIZE * sizeof(float), hipMemcpyDeviceToDevice);
	}

	update_velocity <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, v, p);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, Boundary::U); 
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (v, Boundary::V);
}

__host__ void advect(float *d, const float *d0, const float *u, const float *v, const float dissipation, Boundary mode)
{
	transport <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (d, d0, u, v, dissipation);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (d, mode);
}

__host__ void vorticity_confine(float *u, float *v, float *curl, const float *u0, const float *v0)
{
	compute_curl <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u0, v0, curl);
	vc_update <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, v, curl, u0, v0);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (u, Boundary::U);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (v, Boundary::V);
}

struct Lab1VideoGenerator::Impl {
	int t = 0;
	float *u, *v, *p;
	float *u0, *v0, *p0;
	float *u_source, *v_source;
	float *u_temp, *v_temp;

	float *div;
	float *curl;

	float *d, *d0;
	float *d_source;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
	hipMalloc(&(impl->u), SIZE * sizeof(float));
	hipMalloc(&(impl->v), SIZE * sizeof(float));
	hipMalloc(&(impl->p), SIZE * sizeof(float));
	hipMalloc(&(impl->d), SIZE * sizeof(float));

	hipMalloc(&(impl->u0), SIZE * sizeof(float));
	hipMalloc(&(impl->v0), SIZE * sizeof(float));
	hipMalloc(&(impl->p0), SIZE * sizeof(float));
	hipMalloc(&(impl->d0), SIZE * sizeof(float));

	hipMalloc(&(impl->u_source), SIZE * sizeof(float));
	hipMalloc(&(impl->v_source), SIZE * sizeof(float));
	hipMalloc(&(impl->d_source), SIZE * sizeof(float));

	hipMalloc(&(impl->div), SIZE * sizeof(float));
	hipMalloc(&(impl->curl), SIZE * sizeof(float));

	hipMalloc(&(impl->u_temp), SIZE * sizeof(float));
	hipMalloc(&(impl->v_temp), SIZE * sizeof(float));

	hipMemset(impl->u0, 0, SIZE * sizeof(float));
	hipMemset(impl->v0, 0, SIZE * sizeof(float));
	hipMemset(impl->d0, 0, SIZE * sizeof(float));

	hipMemset(impl->u_source, 0, SIZE * sizeof(float));
	hipMemset(impl->v_source, 0, SIZE * sizeof(float));
	hipMemset(impl->d_source, 0, SIZE * sizeof(float));

	init_source <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->u_source, impl->v_source, impl->d_source);
}

Lab1VideoGenerator::~Lab1VideoGenerator() {
	hipFree(impl->u), hipFree(impl->u0), hipFree(impl->u_source), hipFree(impl->u_temp);
	hipFree(impl->v), hipFree(impl->v0), hipFree(impl->v_source), hipFree(impl->v_temp);
	hipFree(impl->p), hipFree(impl->p0);
	hipFree(impl->d), hipFree(impl->d0), hipFree(impl->d_source);

	hipFree(impl->div);
	hipFree(impl->curl);
}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) 
{
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

void Lab1VideoGenerator::Generate(uint8_t *yuv) 
{
	/* velocity step */
	add_force <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->u0, impl->u_source);
	add_force <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->v0, impl->v_source);

	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->u0, Boundary::U);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->v0, Boundary::V);

	// vorticity confinement
	vorticity_confine(impl->u_temp, impl->v_temp, impl->curl, impl->u0, impl->v0);
	add_force <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->u0, impl->u_temp);
	add_force <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->v0, impl->v_temp);

	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->u0, Boundary::U);
	set_boundary <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->v0, Boundary::V);

	project(impl->u0, impl->v0, impl->p, impl->p0, impl->div);

	// transport
	advect(impl->u, impl->u0, impl->u0, impl->v0, 1.0, Boundary::U);
	advect(impl->v, impl->v0, impl->u0, impl->v0, 1.0, Boundary::V);

	project(impl->u, impl->v, impl->p, impl->p0, impl->div);

	/* end velocity step */

	/* scalar step */
	add_force <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->d0, impl->d_source);

	advect(impl->d, impl->d0, impl->u, impl->v, 0.995, Boundary::D);

	/* end scalar step */

	hipDeviceSynchronize();

	hipMemcpy(impl->u0, impl->u, SIZE * sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(impl->v0, impl->v, SIZE * sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(impl->d0, impl->d, SIZE * sizeof(float), hipMemcpyDeviceToDevice);

	// copy to frame
	density_to_yuv <<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>> (impl->d, yuv);

	++(impl->t);
}
